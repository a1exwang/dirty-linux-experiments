#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <nccl.h>
#include "allreduce.h"
#include <mpi.h>

struct NcclContextImpl {
  hipStream_t stream;
  hipblasHandle_t cublas_handle;
  ncclComm_t comm;
  ncclUniqueId comm_id;
};

using namespace std;

void NcclContext::ncclInit() {
  hipSetDevice(device_id);
  cout << "setDevice: " << device_id << endl;

  int priority_high, priority_low;
  hipDeviceGetStreamPriorityRange(&priority_low, &priority_high);
  // Create stream with highest available priorities
  hipError_t e = hipStreamCreateWithPriority(&this->impl->stream, hipStreamNonBlocking, priority_high);
//  hipError_t e = hipStreamCreate(&this->impl->stream);
  if (e != hipSuccess) {
    cerr << "cudaStreamCreateFailed: " << hipGetErrorString(e) << endl;
    return;
  }
  cout << "streamCreate: " << impl->stream << endl;

  ncclGetUniqueId(&this->impl->comm_id);
  MPI_Bcast(&this->impl->comm_id, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD);
  cout << this->impl->comm << " " << proc_size << " " << rank;
  ncclCommInitRank(&this->impl->comm, proc_size, this->impl->comm_id, rank);

  //int priority_high, priority_low;
  //hipDeviceGetStreamPriorityRange(&priority_low, &priority_high);
  // Create stream with highest available priorities
  //hipStreamCreateWithPriority(&this->impl->stream, hipStreamNonBlocking, priority_high);
}

void NcclContext::allReduce(const void *sendbuf, void *recvbuf, size_t count) {
  ncclResult_t r = ncclAllReduce(sendbuf, recvbuf, count, ncclFloat, ncclSum, this->impl->comm, this->impl->stream);
  if (r != ncclSuccess) {
    std::cerr << "ncclError: " << ncclGetErrorString(r) << std::endl;
  }
}

NcclContext::NcclContext(int proc_size, int rank) :impl(new NcclContextImpl), proc_size(proc_size), rank(rank), device_id(rank) {}

void *NcclContext::alloc(uint64_t size) {
  void *ret;
  hipError_t r = hipMallocManaged(&ret, size);
  if (r != hipSuccess) {
    cerr << "hipMallocManaged Failed: " << hipGetErrorString(r) << endl;
    return 0;
  }
  return ret;
}

void NcclContext::sync() {
  hipStreamSynchronize(this->impl->stream);
}
