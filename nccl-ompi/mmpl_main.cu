#include <mmpl.hpp>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <iostream>
#include <chrono>

using namespace std;

int main(int argc, char **argv) {
    int p_rank, p_size;

    MMPL_Init(NULL, NULL, &p_rank, &p_size, false);
    MMPL_SetDevice(p_rank);

    size_t size = 3079000;
    void *buf;
    hipError_t r = hipMallocManaged(&buf, size * sizeof(float));
    if (r != hipSuccess) {
        cerr << "hipMallocManaged Failed: " << hipGetErrorString(r) << endl;
        return 1;
    }

    MPI_Barrier(MPI_COMM_WORLD);

    for (int i = 0; ; i++) {
        auto t0 = std::chrono::high_resolution_clock::now();
        MMPL_Allreduce_gpu(buf, buf, size, 0, MMPL_FLOAT);
        auto t1 = std::chrono::high_resolution_clock::now();
        cout << "Allreduce rank=:" << p_rank << " i=" << i << " t=" << (t1-t0).count()/1e6 << "ms" << endl;
    }

//    MMPL_Finalize();
}